#include "../../imp/common/book.h"

// I am not sure how this works.
// for example, you can ask for a device with a compute capability higher
// than the system is capable of and still get back an inappropriate device.
int main() {
    hipDeviceProp_t prop{};

    int dev;

    HANDLE_ERROR(hipGetDevice(&dev));

    printf("The ID of the current cuda device: %d\n", dev);

    memset(&prop, 0, sizeof(hipDeviceProp_t));
//    prop.major = 3;
//    prop.minor = 1;
    prop.multiProcessorCount = 8;

    printf("%d\n", prop.multiProcessorCount);
    hipChooseDevice(&dev, &prop );
//    printf("ID of CUDA device closest to revision %d.%d: %d\n",
//           prop.major,
//           prop.minor,
//           dev);

    printf("Looking for GPU with TextureAlignment of 1024\n");
    printf("Selected GPU: %d\n", dev);
    hipGetDeviceProperties(&prop, dev);
    printf("Selected GPU name: %s\n", prop.name);

    HANDLE_ERROR(hipSetDevice(dev));
    return 0;
}

