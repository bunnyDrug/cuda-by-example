#include "../../imp/common/book.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main() {
    hipDeviceProp_t prop{};

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i=0; i<count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

        // do something with the device properties
        printf("   --- General Information for device %d ---\n", i);
        printf("Name: %s\n",                    prop.name);
        printf("Compute capability: %d.%d\n",   prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);
        printf("Device copy overlap: %s\n", (prop.deviceOverlap ? ("Enabled") : ("Disabled")));
        printf("Kernel execution timeout: %s\n", (prop.kernelExecTimeoutEnabled ? ("Enabled") : ("Disabled")));
        
        printf("   --- Memory Information for device %d ---\n", i);
        printf("Total global memory: %ld\n", prop.totalGlobalMem);
        printf("Total constant memory: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem p/mp: %ld\n", prop.sharedMemPerMultiprocessor);
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                prop.maxThreadsDim[0],
                prop.maxThreadsDim[1],
                prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                prop.maxGridSize[0],
                prop.maxGridSize[1],
                prop.maxGridSize[2] );
        printf( "\n" );
    }
}

