#include "hip/hip_runtime.h"
#include <stdio.h>
#include "../../imp/common/book.h"

__global__ void add ( int a, int b, int *c ) {
    *c = a + b;
}

int main() {
    int c;
    int *dev_c;

    // allocate gpu memory to the size of int - pointer in dev_c
    HANDLE_ERROR(hipMalloc( &dev_c, sizeof(int) ) );

    add<<<1,1>>>( 2, 7, dev_c);

    HANDLE_ERROR(hipMemcpy( &c, dev_c, sizeof(int), hipMemcpyDeviceToHost) );

    printf( "2 + 7 = %d\n", c );
    hipFree( dev_c );

    return 0;
}

