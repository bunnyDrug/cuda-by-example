#include "../../imp/common/book.h"


int main() {
    hipDeviceProp_t prop{};

    int count;
    HANDLE_ERROR(hipGetDeviceCount(&count));
    for (int i=0; i<count; i++) {
        HANDLE_ERROR(hipGetDeviceProperties(&prop, i));

        // do something with the device properties
        printf("   --- General Information for device %d ---\n", i);
        printf("Name: %s\n", prop.name);
        printf("Compute capability: %d.%d\n", prop.major, prop.minor);
        printf("Clock rate: %d\n", prop.clockRate);

        printf("Device copy overlap:  ");
        if (prop.deviceOverlap) {
            printf("Enabled\n");
        } else {
            printf("Disabled\n");
        }

        printf("Kernel execution timeout: ");
        if (prop.kernelExecTimeoutEnabled) {
            printf("Enabled\n");
        } else {
            printf("Disabled\n");
        }

        printf("   --- Memory Information for device %d ---\n", i);
        printf("Total global memory: %ld\n", prop.totalGlobalMem);
        printf("Total constant memory: %ld\n", prop.totalConstMem);
        printf("Max mem pitch: %ld\n", prop.memPitch);
        printf("Texture Alignment %ld\n", prop.textureAlignment);

        printf("   --- MP Information for device %d ---\n", i);
        printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
        printf("Shared mem per mp: %d\n", prop.sharedMemPerMultiprocessor);
    }
}

