#include "hip/hip_runtime.h"
#include "../../imp/common/book.h"
#define N 10000

// lets look at this in detail:
// typically you would write something a lot simpler here but the code is a
// suggestion at a way to parallelize the code on a system with multiple CPUs
// or CPU cores.
__global__ void add (int *a, int *b, int *c) {
    int tid = blockIdx.x;   // handle the data at this index
    if (tid < N) { // Thread ID < kernel launch.
        c[tid] = a[tid] + b[tid];
    }
}
// You could have threads execute adds on odd and even values in the array
// but this would require more code and infrastructure.

int main() {
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++){
        a[i] = -i;
        b[i] = i * i;
    }

    hipMalloc(&dev_a, N * sizeof(int));
    hipMalloc(&dev_b, N * sizeof(int));
    hipMalloc(&dev_c, N * sizeof(int));

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    // N blocks running on the GPU
    add<<<N, 1>>>(dev_a, dev_b, dev_c);

    // copy device c back to host
    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

    // display the results.
    for (int i=0; i<N; i++) {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
